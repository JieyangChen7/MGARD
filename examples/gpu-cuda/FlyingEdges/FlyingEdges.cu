#include "hip/hip_runtime.h"


// #include "/home/jieyang/dev/MGARD/include/cuda/FlyingEdges.hpp"

#include <vtkm/io/writer/VTKDataSetWriter.h>
#include <vtkm/io/reader/VTKDataSetReader.h>
#include <vtkm/cont/Initialize.h>
#include <vtkm/cont/DataSet.h>
#include <vtkm/cont/DataSetBuilderUniform.h>
#include <vtkm/cont/Invoker.h>
#include <vtkm/cont/ArrayHandle.h>
#include <vtkm/cont/DataSetFieldAdd.h>
#include <vtkm/cont/ColorTable.h>
#include <vtkm/cont/ArrayCopy.h>
#include <vtkm/cont/ArrayHandlePermutation.h>
#include <vtkm/cont/ArrayHandleUniformPointCoordinates.h>

#include <vtkm/worklet/contour/CommonState.h>
#include <vtkm/worklet/contour/FieldPropagation.h>
#include <vtkm/worklet/contour/FlyingEdges.h>
#include <vtkm/worklet/contour/MarchingCells.h>
#include <vtkm/worklet/SurfaceNormals.h>

#include <vtkm/filter/PolicyBase.h>
#include <vtkm/filter/FieldMetadata.h>
#include <vtkm/filter/FilterDataSetWithField.h>
#include <vtkm/filter/MapFieldPermutation.h>
#include <vtkm/filter/Contour.h>

#include <vtkm/rendering/MapperRayTracer.h>
#include <vtkm/rendering/MapperWireframer.h>
#include <vtkm/rendering/CanvasRayTracer.h>
#include <vtkm/rendering/Actor.h>
#include <vtkm/rendering/Scene.h>
#include <vtkm/rendering/View3D.h>

#include "FlyingEdges.hpp"


bool require_arg(int argc, char *argv[], std::string option) {
  for (int i = 0; i < argc; i++) {
    if (option.compare(std::string(argv[i])) == 0) {
      return true;
    }
  }
  exit(-1);
}

template <typename T> 
size_t readfile(const char *input_file, T *&in_buff) {
  std::cout << mgard_cuda::log::log_info << "Loading file: " << input_file
            << "\n";

  FILE *pFile;
  pFile = fopen(input_file, "rb");
  if (pFile == NULL) {
    std::cout << mgard_cuda::log::log_err << "file open error!\n";
    exit(1);
  }
  fseek(pFile, 0, SEEK_END);
  size_t lSize = ftell(pFile);
  rewind(pFile);
  in_buff = (T *)malloc(lSize);
  lSize = fread(in_buff, 1, lSize, pFile);
  fclose(pFile);
  // min_max(lSize/sizeof(T), in_buff);
  return lSize;
}

std::string get_arg(int argc, char *argv[], std::string option) {
  if (require_arg(argc, argv, option)) {
    for (int i = 0; i < argc; i++) {
      if (option.compare(std::string(argv[i])) == 0) {
        return std::string(argv[i + 1]);
      }
    }
  }
  return std::string("");
}


int get_arg_int(int argc, char *argv[], std::string option) {
  if (require_arg(argc, argv, option)) {
    std::string arg;
    int i;
    for (i = 0; i < argc; i++) {
      if (option.compare(std::string(argv[i])) == 0) {
        arg = std::string(argv[i + 1]);
      }
    }
    try {
      int d = std::stoi(arg);
      return d;
    } catch (std::invalid_argument const &e) {
      exit(-1);
    }
  }
  return 0;
}

std::vector<mgard_cuda::SIZE> get_arg_dims(int argc, char *argv[],
                                           std::string option) {
  std::vector<mgard_cuda::SIZE> shape;
  if (require_arg(argc, argv, option)) {
    std::string arg;
    int arg_idx = 0;
    for (int i = 0; i < argc; i++) {
      if (option.compare(std::string(argv[i])) == 0) {
        arg = std::string(argv[i + 1]);
        arg_idx = i + 1;
      }
    }
    try {
      int d = std::stoi(arg);
      for (int i = 0; i < d; i++) {
        shape.push_back(std::stoi(argv[arg_idx + 1 + i]));
      }
      return shape;
    } catch (std::invalid_argument const &e) {
      exit(-1);
    }
  }
  return shape;
}

double get_arg_double(int argc, char *argv[], std::string option) {
  if (require_arg(argc, argv, option)) {
    std::string arg;
    int i;
    for (i = 0; i < argc; i++) {
      if (option.compare(std::string(argv[i])) == 0) {
        arg = std::string(argv[i + 1]);
      }
    }
    try {
      double d = std::stod(arg);
      return d;
    } catch (std::invalid_argument const &e) {
      exit(-1);
    }
  }
  return 0;
}

template <typename T>
void test_vtkm(int argc, char *argv[], std::vector<mgard_cuda::SIZE> shape, T *original_data, T iso_value) {
  // vtkm::cont::InitializeOptions options = vtkm::cont::InitializeOptions::
  vtkm::cont::Initialize(argc, argv);
  vtkm::cont::RuntimeDeviceTracker& deviceTracker = vtkm::cont::GetRuntimeDeviceTracker();
  deviceTracker.ForceDevice( vtkm::cont::DeviceAdapterTagCuda());
  vtkm::Id3 dims(shape[0], shape[1], shape[2]);
  vtkm::Id3 org(0, 0, 0);
  vtkm::Id3 spc(1, 1, 1);

  size_t original_size = 1;
  for (mgard_cuda::DIM i = 0; i < shape.size(); i++)
    original_size *= shape[i];


  vtkm::cont::DataSet inputDataSet;
  vtkm::cont::DataSetBuilderUniform dataSetBuilderUniform;
  vtkm::cont::DataSetFieldAdd dsf;
  inputDataSet = dataSetBuilderUniform.Create(dims, org, spc);
  std::vector<T> vec_data(original_data, original_data+original_size);
  dsf.AddPointField(inputDataSet, "v", vec_data);

  vtkm::filter::Contour contour;
  contour.SetGenerateNormals(true);
  contour.SetMergeDuplicatePoints(true);
  contour.SetNumberOfIsoValues(1);
  contour.SetIsoValue(0, iso_value);
  contour.SetActiveField("v");

  vtkm::cont::DataSet ds_from_mc = contour.Execute(inputDataSet);
}


template <typename T>
void test_mine(std::vector<mgard_cuda::SIZE> shape, T *original_data, T iso_value) {
  mgard_cuda::Array<3, T, mgard_cuda::CUDA> v({shape[2], shape[1], shape[0]});
  v.loadData(original_data);

  mgard_cuda::Array<1, mgard_cuda::SIZE, mgard_cuda::CUDA> Triangles;
  mgard_cuda::Array<1, T, mgard_cuda::CUDA> Points;

  mgard_cuda::FlyingEdges<T, mgard_cuda::CUDA>().Execute(shape[2], shape[1], shape[0],
                                     mgard_cuda::SubArray<3, T, mgard_cuda::CUDA>(v),
                                     iso_value, Triangles, Points, 0);

  mgard_cuda::PrintSubarray("Triangles", mgard_cuda::SubArray(Triangles));
  mgard_cuda::PrintSubarray("Points", mgard_cuda::SubArray(Points));

  
}

int main(int argc, char *argv[]) {

  std::cout << "start\n";
  std::string input_file = get_arg(argc, argv, "-i");
  mgard_cuda::DIM D = get_arg_int(argc, argv, "-n");
  std::vector<mgard_cuda::SIZE> shape = get_arg_dims(argc, argv, "-n");

  size_t original_size = 1;
  for (mgard_cuda::DIM i = 0; i < D; i++)
    original_size *= shape[i];
  float *original_data;
  size_t in_size = 0;

  if (std::string(input_file).compare("random") == 0) {
    std::cout << "generating data...";
    in_size = original_size * sizeof(float);
    original_data = new float[original_size];
    for (size_t i = 0; i < shape[2]; i++){
      for (size_t j = 0; j < shape[1]; j++){
        for (size_t k = 0; k < shape[0]; k++){
          original_data[i*shape[1]*shape[0]+j*shape[0]+k] = j;
        }
      }
    }
    std::cout << "Done\n";
  } else {
    in_size = readfile(input_file.c_str(), original_data);
  }
  if (in_size != original_size * sizeof(float)) {
    std::cout << mgard_cuda::log::log_err << "input file size mismatch" << in_size << "vs." << original_size * sizeof(float) << "!\n";
  }

  float iso_value = 1.5;
  std::cout << "test_vtkm\n";
  test_vtkm(argc, argv, shape, original_data, iso_value);
  std::cout << "test_mine\n";
  test_mine(shape, original_data, iso_value);


  


}