#include "hip/hip_runtime.h"
#include "mgard-x/MDRHighLevel/qoi_kernel.hpp"
#include <iostream>

namespace mgard::MDR {

__device__ bool check_flag(int *flag) {
  return atomicAdd(flag, 0);
}
__device__ void raise_flag(int *flag) {
  atomicExch(flag, 1);
}

template <class T>
void V_TOT_computation(const T *Vx, const T *Vy, const T *Vz, T *V_TOT, size_t n){
  dim3 block(BLOCK_SIZE);
  dim3 grid((n + BLOCK_SIZE - 1) / BLOCK_SIZE);
  compute_V_TOT<T><<<grid, block>>>(Vx, Vy, Vz, V_TOT, n);
  hipDeviceSynchronize();
}

template <class T>
void V_TOT_computation(const T *Vx, T *V_TOT, size_t n){
  dim3 block(BLOCK_SIZE);
  dim3 grid((n + BLOCK_SIZE - 1) / BLOCK_SIZE);
  compute_V_TOT<T><<<grid, block>>>(Vx, V_TOT, n);
  hipDeviceSynchronize();
}

template <class T>
bool V_TOT_error_estimation(const T *Vx, const T *Vy, const T *Vz, size_t n, double eb_Vx, double eb_Vy, double eb_Vz, double tolerance){
  // std::cout << "From CUDA: eb_Vx: " << eb_Vx << ", eb_Vy: " << eb_Vy << ", eb_Vz: " << eb_Vz << ", requested QoI error: " << tolerance << std::endl;
  dim3 block(BLOCK_SIZE);
  dim3 grid((n + BLOCK_SIZE - 1) / BLOCK_SIZE);
  int tolerance_exceed_flag_h;
  int *tolerance_exceed_flag_d;
  hipMalloc((void**) &tolerance_exceed_flag_d, sizeof(int));
  hipMemset(tolerance_exceed_flag_d, 0, sizeof(int));
  estimate_error_V_TOT<T><<<grid, block>>>(Vx, Vy, Vz, n, eb_Vx, eb_Vy, eb_Vz, tolerance, tolerance_exceed_flag_d);
  hipDeviceSynchronize();
  hipMemcpy(&tolerance_exceed_flag_h, tolerance_exceed_flag_d, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(tolerance_exceed_flag_d);
  // std::cout << "From CUDA: tolerance_exceed_flag_h = " << tolerance_exceed_flag_h << std::endl;
  if (tolerance_exceed_flag_h == 0){
    return false;
  }
  else{
    return true;
  }
}

template <class T>
void V_TOT_error_estimation(const T *Vx, size_t n, double eb_Vx, double eb_Vy, double eb_Vz, double tolerance){
  dim3 block(BLOCK_SIZE);
  dim3 grid((n + BLOCK_SIZE - 1) / BLOCK_SIZE);
  // estimate_error_V_TOT<T><<<grid, block>>>(Vx, n, ebs, error_est_V_TOT, error_V_TOT, V_TOT_ori);
  hipDeviceSynchronize();
}

template void V_TOT_computation<float>(const float*, const float*, const float*, float*, size_t);
template void V_TOT_computation<float>(const float*, float*, size_t);
template bool V_TOT_error_estimation<float>(const float*, const float*, const float*, size_t, double, double, double, double);
template void V_TOT_error_estimation<float>(const float*, size_t, double, double, double, double);

template void V_TOT_computation<double>(const double*, const double*, const double*, double*, size_t);
template void V_TOT_computation<double>(const double*, double*, size_t);
template bool V_TOT_error_estimation<double>(const double*, const double*, const double*, size_t, double, double, double, double);
template void V_TOT_error_estimation<double>(const double*, size_t, double, double, double, double);
} // namespace mgard::MDR
